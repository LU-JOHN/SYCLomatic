
#include <hip/hip_runtime.h>
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none --assume-nd-range-dim=1  -out-root %T/cu_jit %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cu_jit/cu_jit.dp.cpp --match-full-lines %s

#define CU_JIT_NOT_A_CUDA_OPTION 1241

int main() {
  int a[40];
  int CUvar;

//CHECK:a[0] = {{[0-9]+}};
//CHECK-NEXT:a[1] = {{[0-9]+}};
//CHECK-NEXT:a[2] = {{[0-9]+}};
//CHECK-NEXT:a[3] = {{[0-9]+}};
//CHECK-NEXT:a[4] = {{[0-9]+}};
//CHECK-NEXT:a[5] = {{[0-9]+}};
//CHECK-NEXT:a[6] = {{[0-9]+}};
//CHECK-NEXT:a[7] = {{[0-9]+}};
//CHECK-NEXT:a[8] = {{[0-9]+}};
//CHECK-NEXT:a[9] = {{[0-9]+}};
//CHECK-NEXT:a[10] = {{[0-9]+}};
//CHECK-NEXT:a[11] = {{[0-9]+}};
//CHECK-NEXT:a[12] = {{[0-9]+}};
//CHECK-NEXT:a[13] = {{[0-9]+}};
//CHECK-NEXT:a[14] = {{[0-9]+}};
//CHECK-NEXT:a[15] = {{[0-9]+}};
//CHECK-NEXT:a[16] = {{[0-9]+}};
//CHECK-NEXT:a[17] = {{[0-9]+}};
//CHECK-NEXT:a[18] = {{[0-9]+}};
//CHECK-NEXT:a[19] = {{[0-9]+}};
//CHECK-NEXT:a[20] = {{[0-9]+}};
//CHECK-NEXT:a[21] = {{[0-9]+}};
//CHECK-NEXT:a[22] = {{[0-9]+}};
//CHECK-NEXT:a[23] = {{[0-9]+}};
//CHECK-NEXT:a[24] = {{[0-9]+}};
//CHECK-NEXT:a[25] = {{[0-9]+}};
//CHECK-NEXT:a[26] = {{[0-9]+}};
//CHECK-NEXT:a[27] = CU_JIT_NOT_A_CUDA_OPTION;

  a[0] = hipJitOptionMaxRegisters;
  a[1] = hipJitOptionThreadsPerBlock;
  a[2] = hipJitOptionWallTime;
  a[3] = hipJitOptionInfoLogBuffer;
  a[4] = hipJitOptionInfoLogBufferSizeBytes;
  a[5] = hipJitOptionErrorLogBuffer;
  a[6] = hipJitOptionErrorLogBufferSizeBytes;
  a[7] = hipJitOptionOptimizationLevel;
  a[8] = hipJitOptionTargetFromContext;
  a[9] = hipJitOptionTarget;
  a[10] = hipJitOptionFallbackStrategy;
  a[11] = hipJitOptionGenerateDebugInfo;
  a[12] = hipJitOptionLogVerbose;
  a[13] = hipJitOptionGenerateLineInfo;
  a[14] = hipJitOptionCacheMode;
  a[15] = hipJitOptionSm3xOpt;
  a[16] = hipJitOptionFastCompile;
  a[17] = hipJitOptionNumOptions;
  a[18] = CU_JIT_CACHE_OPTION_NONE;
  a[19] = CU_JIT_CACHE_OPTION_CG;
  a[20] = CU_JIT_CACHE_OPTION_CA;
  a[21] = hipJitInputCubin;
  a[22] = hipJitInputPtx;
  a[23] = hipJitInputFatBinary;
  a[24] = hipJitInputObject;
  a[25] = hipJitInputLibrary;
  a[26] = hipJitNumLegacyInputTypes;
  a[27] = CU_JIT_NOT_A_CUDA_OPTION;
}