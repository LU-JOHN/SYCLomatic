
#include <hip/hip_runtime.h>
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none --assume-nd-range-dim=1  -out-root %T/cu_jit %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cu_jit/cu_jit.dp.cpp --match-full-lines %s

#define CU_JIT_NOT_A_CUDA_OPTION 1241

int main() {
  int a[40];
  int CUvar;

//CHECK:a[0] = {{[0-9]+}};
//CHECK-NEXT:a[1] = {{[0-9]+}};
//CHECK-NEXT:a[2] = {{[0-9]+}};
//CHECK-NEXT:a[3] = {{[0-9]+}};
//CHECK-NEXT:a[4] = {{[0-9]+}};
//CHECK-NEXT:a[5] = {{[0-9]+}};
//CHECK-NEXT:a[6] = {{[0-9]+}};
//CHECK-NEXT:a[7] = {{[0-9]+}};
//CHECK-NEXT:a[8] = {{[0-9]+}};
//CHECK-NEXT:a[9] = {{[0-9]+}};
//CHECK-NEXT:a[10] = {{[0-9]+}};
//CHECK-NEXT:a[11] = {{[0-9]+}};
//CHECK-NEXT:a[12] = {{[0-9]+}};
//CHECK-NEXT:a[13] = {{[0-9]+}};
//CHECK-NEXT:a[14] = {{[0-9]+}};
//CHECK-NEXT:a[15] = {{[0-9]+}};
//CHECK-NEXT:a[16] = {{[0-9]+}};
//CHECK-NEXT:a[17] = {{[0-9]+}};
//CHECK-NEXT:a[18] = {{[0-9]+}};
//CHECK-NEXT:a[19] = {{[0-9]+}};
//CHECK-NEXT:a[20] = {{[0-9]+}};
//CHECK-NEXT:a[21] = {{[0-9]+}};
//CHECK-NEXT:a[22] = {{[0-9]+}};
//CHECK-NEXT:a[23] = {{[0-9]+}};
//CHECK-NEXT:a[24] = {{[0-9]+}};
//CHECK-NEXT:a[25] = {{[0-9]+}};
//CHECK-NEXT:a[26] = {{[0-9]+}};
//CHECK-NEXT:a[27] = {{[0-9]+}};
//CHECK-NEXT:a[28] = {{[0-9]+}};
//CHECK-NEXT:a[29] = {{[0-9]+}};
//CHECK-NEXT:a[30] = {{[0-9]+}};
//CHECK-NEXT:a[31] = {{[0-9]+}};
//CHECK-NEXT:a[32] = {{[0-9]+}};
//CHECK-NEXT:a[33] = {{[0-9]+}};
//CHECK-NEXT:a[34] = {{[0-9]+}};
//CHECK-NEXT:a[35] = {{[0-9]+}};
//CHECK-NEXT:a[36] = CU_JIT_NOT_A_CUDA_OPTION;

  a[0] = hipJitOptionMaxRegisters;
  a[1] = hipJitOptionThreadsPerBlock;
  a[2] = hipJitOptionWallTime;
  a[3] = hipJitOptionInfoLogBuffer;
  a[4] = hipJitOptionInfoLogBufferSizeBytes;
  a[5] = hipJitOptionErrorLogBuffer;
  a[6] = hipJitOptionErrorLogBufferSizeBytes;
  a[7] = hipJitOptionOptimizationLevel;
  a[8] = hipJitOptionTargetFromContext;
  a[9] = hipJitOptionTarget;
  a[10] = hipJitOptionFallbackStrategy;
  a[11] = hipJitOptionGenerateDebugInfo;
  a[12] = hipJitOptionLogVerbose;
  a[13] = hipJitOptionGenerateLineInfo;
  a[14] = hipJitOptionCacheMode;
  a[15] = hipJitOptionSm3xOpt;
  a[16] = hipJitOptionFastCompile;
  a[17] = hipJitOptionGlobalSymbolNames;
  a[18] = hipJitOptionGlobalSymbolAddresses;
  a[19] = hipJitOptionGlobalSymbolCount;
  a[20] = hipJitOptionLto;
  a[21] = hipJitOptionFtz;
  a[22] = hipJitOptionPrecDiv;
  a[23] = hipJitOptionPrecSqrt;
  a[24] = hipJitOptionFma;
  a[25] = hipJitOptionNumOptions;
  a[26] = CU_JIT_CACHE_OPTION_NONE;
  a[27] = CU_JIT_CACHE_OPTION_CG;
  a[28] = CU_JIT_CACHE_OPTION_CA;
  a[29] = hipJitInputCubin;
  a[30] = hipJitInputPtx;
  a[31] = hipJitInputFatBinary;
  a[32] = hipJitInputObject;
  a[33] = hipJitInputLibrary;
  a[34] = hipJitInputNvvm;
  a[35] = hipJitNumLegacyInputTypes;
  a[36] = CU_JIT_NOT_A_CUDA_OPTION;
}